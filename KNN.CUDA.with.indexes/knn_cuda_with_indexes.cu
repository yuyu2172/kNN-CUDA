#include "hip/hip_runtime.h"
/**
  *
  * Date         03/07/2009
  * ====
  *
  * Authors      Vincent Garcia
  * =======      Eric    Debreuve
  *              Michel  Barlaud
  *
  * Description  Given a reference point set and a query point set, the program returns
  * ===========  firts the distance between each query point and its k nearest neighbors in
  *              the reference point set, and second the indexes of these k nearest neighbors.
  *              The computation is performed using the API NVIDIA CUDA.
  *
  * Paper        Fast k nearest neighbor search using GPU
  * =====
  *
  * BibTeX       @INPROCEEDINGS{2008_garcia_cvgpu,
  * ======         author = {V. Garcia and E. Debreuve and M. Barlaud},
  *                title = {Fast k nearest neighbor search using GPU},
  *                booktitle = {CVPR Workshop on Computer Vision on GPU},
  *                year = {2008},
  *                address = {Anchorage, Alaska, USA},
  *                month = {June}
  *              }
  *
  */


// If the code is used in Matlab, set MATLAB_CODE to 1. Otherwise, set MATLAB_CODE to 0.
#define MATLAB_CODE 0  


// Includes
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"
#if MATLAB_CODE == 1
	#include "mex.h"
#else
	#include <time.h>
#endif


// Constants used by the program
#define MAX_PITCH_VALUE_IN_BYTES       262144
#define MAX_TEXTURE_WIDTH_IN_BYTES     65536
#define MAX_TEXTURE_HEIGHT_IN_BYTES    32768
#define MAX_PART_OF_FREE_MEMORY_USED   0.9
#define BLOCK_DIM                      16


// Texture containing the reference points (if it is possible)
texture<float, 2, hipReadModeElementType> texA;



//-----------------------------------------------------------------------------------------------//
//                                            KERNELS                                            //
//-----------------------------------------------------------------------------------------------//



/**
  * Computes the distance between two matrix A (reference points) and
  * B (query points) containing respectively wA and wB points.
  * The matrix A is a texture.
  *
  * @param wA    width of the matrix A = number of points in A
  * @param B     pointer on the matrix B
  * @param wB    width of the matrix B = number of points in B
  * @param pB    pitch of matrix B given in number of columns
  * @param dim   dimension of points = height of matrices A and B
  * @param AB    pointer on the matrix containing the wA*wB distances computed
  */
__global__ void cuComputeDistanceTexture(int wA, float * B, int wB, int pB, int dim, float* AB){
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if ( xIndex<wB && yIndex<wA ){
        float ssd = 0;
        for (int i=0; i<dim; i++){
            float tmp  = tex2D(texA, (float)yIndex, (float)i) - B[ i * pB + xIndex ];
            ssd += tmp * tmp;
        }
        AB[yIndex * pB + xIndex] = ssd;
    }
}


/**
  * Computes the distance between two matrix A (reference points) and
  * B (query points) containing respectively wA and wB points.
  *
  * @param A     pointer on the matrix A
  * @param wA    width of the matrix A = number of points in A
  * @param pA    pitch of matrix A given in number of columns
  * @param B     pointer on the matrix B
  * @param wB    width of the matrix B = number of points in B
  * @param pB    pitch of matrix B given in number of columns
  * @param dim   dimension of points = height of matrices A and B
  * @param AB    pointer on the matrix containing the wA*wB distances computed
  */
__global__ void cuComputeDistanceGlobal( float* A, int wA, int pA, float* B, int wB, int pB, int dim,  float* AB){

	// Declaration of the shared memory arrays As and Bs used to store the sub-matrix of A and B
	__shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
	__shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];
    
    // Sub-matrix of A (begin, step, end) and Sub-matrix of B (begin, step)
    __shared__ int begin_A;
    __shared__ int begin_B;
    __shared__ int step_A;
    __shared__ int step_B;
    __shared__ int end_A;
	
    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
	
	// Other variables
	float tmp;
    float ssd = 0;
	
    // Loop parameters
    begin_A = BLOCK_DIM * blockIdx.y;
    begin_B = BLOCK_DIM * blockIdx.x;
    step_A  = BLOCK_DIM * pA;
    step_B  = BLOCK_DIM * pB;
    end_A   = begin_A + (dim-1) * pA;
    
    // Conditions
	int cond0 = (begin_A + tx < wA); // used to write in shared memory
    int cond1 = (begin_B + tx < wB); // used to write in shared memory & to computations and to write in output matrix
    int cond2 = (begin_A + ty < wA); // used to computations and to write in output matrix
    
    // Loop over all the sub-matrices of A and B required to compute the block sub-matrix
    for (int a = begin_A, b = begin_B; a <= end_A; a += step_A, b += step_B) {
        
        // Load the matrices from device memory to shared memory; each thread loads one element of each matrix
        if (a/pA + ty < dim){
            shared_A[ty][tx] = (cond0)? A[a + pA * ty + tx] : 0;
            shared_B[ty][tx] = (cond1)? B[b + pB * ty + tx] : 0;
        }
        else{
            shared_A[ty][tx] = 0;
            shared_B[ty][tx] = 0;
        }
        
        // Synchronize to make sure the matrices are loaded
        __syncthreads();
        
        // Compute the difference between the two matrixes; each thread computes one element of the block sub-matrix
        if (cond2 && cond1){
            for (int k = 0; k < BLOCK_DIM; ++k){
				tmp = shared_A[k][ty] - shared_B[k][tx];
                ssd += tmp*tmp;
			}
        }
        
        // Synchronize to make sure that the preceding computation is done before loading two new sub-matrices of A and B in the next iteration
        __syncthreads();
    }
    
    // Write the block sub-matrix to device memory; each thread writes one element
    if (cond2 && cond1)
        AB[ (begin_A + ty) * pB + begin_B + tx ] = ssd;
}



/**
  * Gathers k-th smallest distances for each column of the distance matrix in the top.
  *
  * @param dist        distance matrix
  * @param dist_pitch  pitch of the distance matrix given in number of columns
  * @param ind         index matrix
  * @param ind_pitch   pitch of the index matrix given in number of columns
  * @param width       width of the distance matrix and of the index matrix
  * @param height      height of the distance matrix and of the index matrix
  * @param k           number of neighbors to consider
  */
__global__ void cuInsertionSort(float *dist, int dist_pitch, int *ind, int ind_pitch, int width, int height, int k){

	// Variables
    int l, i, j;
    float *p_dist;
	int   *p_ind;
    float curr_dist, max_dist;
    int   curr_row,  max_row;
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	
    if (xIndex<width){
        
        // Pointer shift, initialization, and max value
        p_dist   = dist + xIndex;
		p_ind    = ind  + xIndex;
        max_dist = p_dist[0];
        p_ind[0] = 1;
        
        // Part 1 : sort kth firt elementZ
        for (l=1; l<k; l++){
            curr_row  = l * dist_pitch;
			curr_dist = p_dist[curr_row];
			if (curr_dist<max_dist){
                i=l-1;
				for (int a=0; a<l-1; a++){
					if (p_dist[a*dist_pitch]>curr_dist){
						i=a;
						break;
					}
				}
                for (j=l; j>i; j--){
					p_dist[j*dist_pitch] = p_dist[(j-1)*dist_pitch];
					p_ind[j*ind_pitch]   = p_ind[(j-1)*ind_pitch];
                }
				p_dist[i*dist_pitch] = curr_dist;
				p_ind[i*ind_pitch]   = l+1;
			}
			else
				p_ind[l*ind_pitch] = l+1;
			max_dist = p_dist[curr_row];
		}
        
        // Part 2 : insert element in the k-th first lines
        max_row = (k-1)*dist_pitch;
        for (l=k; l<height; l++){
			curr_dist = p_dist[l*dist_pitch];
			if (curr_dist<max_dist){
                i=k-1;
				for (int a=0; a<k-1; a++){
					if (p_dist[a*dist_pitch]>curr_dist){
						i=a;
						break;
					}
				}
                for (j=k-1; j>i; j--){
					p_dist[j*dist_pitch] = p_dist[(j-1)*dist_pitch];
					p_ind[j*ind_pitch]   = p_ind[(j-1)*ind_pitch];
                }
				p_dist[i*dist_pitch] = curr_dist;
				p_ind[i*ind_pitch]   = l+1;
                max_dist             = p_dist[max_row];
            }
        }
    }
}



/**
  * Computes the square root of the first line (width-th first element)
  * of the distance matrix.
  *
  * @param dist    distance matrix
  * @param width   width of the distance matrix
  * @param pitch   pitch of the distance matrix given in number of columns
  * @param k       number of neighbors to consider
  */
__global__ void cuParallelSqrt(float *dist, int width, int pitch, int k){
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if (xIndex<width && yIndex<k)
        dist[yIndex*pitch + xIndex] = sqrt(dist[yIndex*pitch + xIndex]);
}



//-----------------------------------------------------------------------------------------------//
//                                   K-th NEAREST NEIGHBORS                                      //
//-----------------------------------------------------------------------------------------------//



/**
  * Prints the error message return during the memory allocation.
  *
  * @param error        error value return by the memory allocation function
  * @param memorySize   size of memory tried to be allocated
  */
void printErrorMessage(hipError_t error, int memorySize){
    printf("==================================================\n");
    printf("MEMORY ALLOCATION ERROR  : %s\n", hipGetErrorString(error));
    printf("Whished allocated memory : %d\n", memorySize);
    printf("==================================================\n");
#if MATLAB_CODE == 1
    mexErrMsgTxt("CUDA ERROR DURING MEMORY ALLOCATION");
#endif
}



/**
  * K nearest neighbor algorithm
  * - Initialize CUDA
  * - Allocate device memory
  * - Copy point sets (reference and query points) from host to device memory
  * - Compute the distances + indexes to the k nearest neighbors for each query point
  * - Copy distances from device to host memory
  *
  * @param ref_host      reference points ; pointer to linear matrix
  * @param ref_width     number of reference points ; width of the matrix
  * @param query_host    query points ; pointer to linear matrix
  * @param query_width   number of query points ; width of the matrix
  * @param height        dimension of points ; height of the matrices
  * @param k             number of neighbor to consider
  * @param dist_host     distances to k nearest neighbors ; pointer to linear matrix
  * @param dist_host     indexes of the k nearest neighbors ; pointer to linear matrix
  *
  */
void knn(float* ref_host, int ref_width, float* query_host, int query_width, int height, int k, float* dist_host, int* ind_host){
    
    unsigned int size_of_float = sizeof(float);
    unsigned int size_of_int   = sizeof(int);
    
    // Variables
    //float        *query_dev;
    //float        *ref_dev;
    //float        *dist_dev;
    //int          *ind_dev;
    //hipArray    *ref_array;
    //hipError_t  result;
    //size_t       query_pitch;
    //size_t	     query_pitch_in_bytes;
    //size_t       ref_pitch;
    //size_t       ref_pitch_in_bytes;
    //size_t       ind_pitch;
    //size_t       ind_pitch_in_bytes;
    //size_t       max_nb_query_traited;
    //size_t       actual_nb_query_width;
    //unsigned int memory_total;
    //unsigned int memory_free;
 // Variables
float *query_dev;
float *ref_dev;
float *dist_dev;
int *ind_dev;
hipArray *ref_array;
hipError_t result;
size_t query_pitch;
size_t query_pitch_in_bytes;
size_t ref_pitch;
size_t ref_pitch_in_bytes;
size_t       ind_pitch;
size_t       ind_pitch_in_bytes;
unsigned int max_nb_query_traited;
size_t actual_nb_query_width;
size_t memory_total;
size_t  memory_free;   
	
	// Check if we can use texture memory for reference points
    unsigned int use_texture = ( ref_width*size_of_float<=MAX_TEXTURE_WIDTH_IN_BYTES && height*size_of_float<=MAX_TEXTURE_HEIGHT_IN_BYTES );
    
    // CUDA Initialisation
    hipInit(0);
    
    // Check free memory using driver API ; only (MAX_PART_OF_FREE_MEMORY_USED*100)% of memory will be used
    hipCtx_t cuContext;
    hipDevice_t  cuDevice=0;
    hipCtxCreate(&cuContext, 0, cuDevice);
    hipMemGetInfo(&memory_free, &memory_total);
    cuCtxDetach (cuContext);
    
    // Determine maximum number of query that can be treated
    max_nb_query_traited = ( memory_free * MAX_PART_OF_FREE_MEMORY_USED - size_of_float * ref_width*height ) / ( size_of_float * (height + ref_width) + size_of_int * k);
    max_nb_query_traited = min( query_width, (max_nb_query_traited / 16) * 16 );
    
    // Allocation of global memory for query points and for distances
    result = hipMallocPitch( (void **) &query_dev, &query_pitch_in_bytes, max_nb_query_traited * size_of_float, height + ref_width);
    if (result){
        printErrorMessage(result, max_nb_query_traited*size_of_float*(height+ref_width));
        return;
    }
    query_pitch = query_pitch_in_bytes/size_of_float;
    dist_dev    = query_dev + height * query_pitch;
	
    // Allocation of global memory for indexes	
    result = hipMallocPitch( (void **) &ind_dev, &ind_pitch_in_bytes, max_nb_query_traited * size_of_int, k);
	if (result){
        hipFree(query_dev);
        printErrorMessage(result, max_nb_query_traited*size_of_int*k);
        return;
    }
    ind_pitch = ind_pitch_in_bytes/size_of_int;
    
	// Allocation of memory (global or texture) for reference points
    if (use_texture){
	
        // Allocation of texture memory
        hipChannelFormatDesc channelDescA = hipCreateChannelDesc<float>();
        result = hipMallocArray( &ref_array, &channelDescA, ref_width, height );
        if (result){
            printErrorMessage(result, ref_width*height*size_of_float);
            hipFree(ind_dev);
            hipFree(query_dev);
            return;
        }
        hipMemcpyToArray( ref_array, 0, 0, ref_host, ref_width * height * size_of_float, hipMemcpyHostToDevice );
        
        // Set texture parameters and bind texture to array
        texA.addressMode[0] = hipAddressModeClamp;
        texA.addressMode[1] = hipAddressModeClamp;
        texA.filterMode     = hipFilterModePoint;
        texA.normalized     = 0;
        hipBindTextureToArray(texA, ref_array);
		
    }
    else{
	
		// Allocation of global memory
        result = hipMallocPitch( (void **) &ref_dev, &ref_pitch_in_bytes, ref_width * size_of_float, height);
        if (result){
            printErrorMessage(result,  ref_width*size_of_float*height);
            hipFree(ind_dev);
            hipFree(query_dev);
            return;
        }
        ref_pitch = ref_pitch_in_bytes/size_of_float;
        hipMemcpy2D(ref_dev, ref_pitch_in_bytes, ref_host, ref_width*size_of_float,  ref_width*size_of_float, height, hipMemcpyHostToDevice);
    }
    
    // Split queries to fit in GPU memory
    for (int i=0; i<query_width; i+=max_nb_query_traited){
        
		// Number of query points considered
        actual_nb_query_width = min( max_nb_query_traited, query_width-i );
        
        // Copy of part of query actually being treated
        hipMemcpy2D(query_dev, query_pitch_in_bytes, &query_host[i], query_width*size_of_float, actual_nb_query_width*size_of_float, height, hipMemcpyHostToDevice);
        
        // Grids ans threads
        dim3 g_16x16(actual_nb_query_width/16, ref_width/16, 1);
        dim3 t_16x16(16, 16, 1);
        if (actual_nb_query_width%16 != 0) g_16x16.x += 1;
        if (ref_width  %16 != 0) g_16x16.y += 1;
        //
        dim3 g_256x1(actual_nb_query_width/256, 1, 1);
        dim3 t_256x1(256, 1, 1);
        if (actual_nb_query_width%256 != 0) g_256x1.x += 1;
		//
        dim3 g_k_16x16(actual_nb_query_width/16, k/16, 1);
        dim3 t_k_16x16(16, 16, 1);
        if (actual_nb_query_width%16 != 0) g_k_16x16.x += 1;
        if (k  %16 != 0) g_k_16x16.y += 1;
        
        // Kernel 1: Compute all the distances
        if (use_texture)
            cuComputeDistanceTexture<<<g_16x16,t_16x16>>>(ref_width, query_dev, actual_nb_query_width, query_pitch, height, dist_dev);
        else
            cuComputeDistanceGlobal<<<g_16x16,t_16x16>>>(ref_dev, ref_width, ref_pitch, query_dev, actual_nb_query_width, query_pitch, height, dist_dev);
            
        // Kernel 2: Sort each column
        cuInsertionSort<<<g_256x1,t_256x1>>>(dist_dev, query_pitch, ind_dev, ind_pitch, actual_nb_query_width, ref_width, k);
        
        // Kernel 3: Compute square root of k first elements
        cuParallelSqrt<<<g_k_16x16,t_k_16x16>>>(dist_dev, query_width, query_pitch, k);
        
        // Memory copy of output from device to host
		hipMemcpy2D(&dist_host[i], query_width*size_of_float, dist_dev, query_pitch_in_bytes, actual_nb_query_width*size_of_float, k, hipMemcpyDeviceToHost);
        hipMemcpy2D(&ind_host[i],  query_width*size_of_int,   ind_dev,  ind_pitch_in_bytes,   actual_nb_query_width*size_of_int,   k, hipMemcpyDeviceToHost);
    }
    
    // Free memory
    if (use_texture)
        hipFreeArray(ref_array);
    else
        hipFree(ref_dev);
    hipFree(ind_dev);
    hipFree(query_dev);
}



//-----------------------------------------------------------------------------------------------//
//                                MATLAB INTERFACE & C EXAMPLE                                   //
//-----------------------------------------------------------------------------------------------//



#if MATLAB_CODE == 1

/**
  * Interface to use CUDA code in Matlab (gateway routine).
  *
  * @param nlhs  	Number of expected mxArrays (Left Hand Side)
  * @param plhs 	Array of pointers to expected outputs
  * @param nrhs 	Number of inputs (Right Hand Side)
  * @param prhs 	Array of pointers to input data. The input data is read-only and should not be altered by your mexFunction .
  */
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]){
    
	// Variables
    float* ref;
    int    ref_width;
    int    ref_height;
    float* query;
    int    query_width;
    int    query_height;
    float* dist;
	int*   ind;
    int    k;
    
    // Reference points
    ref          = (float *) mxGetData(prhs[0]);
    ref_width    = mxGetM(prhs[0]);
    ref_height   = mxGetN(prhs[0]);
	
	// Query points
    query        = (float *) mxGetData(prhs[1]);
    query_width  = mxGetM(prhs[1]);
    query_height = mxGetN(prhs[1]);
	
	// Number of neighbors to consider
    k            = (int)mxGetScalar(prhs[2]);
    
    // Verification of the reference point and query point sizes
    if (ref_height!=query_height)
        mexErrMsgTxt("Data must have the same dimension");
    if (ref_width*sizeof(float)>MAX_PITCH_VALUE_IN_BYTES)
        mexErrMsgTxt("Reference number is too large for CUDA (Max=65536)");
    if (query_width*sizeof(float)>MAX_PITCH_VALUE_IN_BYTES)
        mexErrMsgTxt("Query number is too large for CUDA (Max=65536)");
    
    // Allocation of output arrays
    dist = (float *) mxGetPr(plhs[0] = mxCreateNumericMatrix(query_width, k, mxSINGLE_CLASS, mxREAL));
    ind  =   (int *) mxGetPr(plhs[1] = mxCreateNumericMatrix(query_width, k, mxINT32_CLASS,  mxREAL));
    
    // Call KNN CUDA
    knn(ref, ref_width, query, query_width, ref_height, k, dist, ind);
}

#else // C code

/**
  * Example of use of kNN search CUDA.
  */
int main(void){
	
    // Variables and parameters
    float* ref;                 // Pointer to reference point array
    float* query;               // Pointer to query point array
    float* dist;                // Pointer to distance array
	int*   ind;                 // Pointer to index array
	int    ref_nb     = 4096;   // Reference point number, max=65535
	int    query_nb   = 4096;   // Query point number,     max=65535
	int    dim        = 32;     // Dimension of points
	int    k          = 20;     // Nearest neighbors to consider
	int    iterations = 100;
	int    i;
	
	// Memory allocation
	ref    = (float *) malloc(ref_nb   * dim * sizeof(float));
	query  = (float *) malloc(query_nb * dim * sizeof(float));
	dist   = (float *) malloc(query_nb * k * sizeof(float));
	ind    = (int *)   malloc(query_nb * k * sizeof(float));
	
	// Init 
	srand(time(NULL));
	for (i=0 ; i<ref_nb   * dim ; i++) ref[i]    = (float)rand() / (float)RAND_MAX;
	for (i=0 ; i<query_nb * dim ; i++) query[i]  = (float)rand() / (float)RAND_MAX;
	
	// Variables for duration evaluation
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float elapsed_time;
	
	// Display informations
	printf("Number of reference points      : %6d\n", ref_nb  );
	printf("Number of query points          : %6d\n", query_nb);
	printf("Dimension of points             : %4d\n", dim     );
	printf("Number of neighbors to consider : %4d\n", k       );
	printf("Processing kNN search           :"                );
	
	// Call kNN search CUDA
	hipEventRecord(start, 0);
	for (i=0; i<iterations; i++)
		knn(ref, ref_nb, query, query_nb, dim, k, dist, ind);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	printf(" done in %f s for %d iterations (%f s by iteration)\n", elapsed_time/1000, iterations, elapsed_time/(iterations*1000));
	
	// Destroy cuda event object and free memory
	hipEventDestroy(start);
	hipEventDestroy(stop);
	free(ind);
	free(dist);
	free(query);
	free(ref);
}

#endif
